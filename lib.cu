#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "lib.h"

__global__ void sum_gpu_kernel(int n, const float* a, const float* b,
                               float* c) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

void sum_gpu(int n, const float* a, const float* b, float* c) {
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, sizeof(float) * n);
  hipMalloc(&d_b, sizeof(float) * n);
  hipMalloc(&d_c, sizeof(float) * n);

  hipMemcpy(d_a, a, sizeof(float) * n, hipMemcpyDefault);
  hipMemcpy(d_b, b, sizeof(float) * n, hipMemcpyDefault);

  sum_gpu_kernel<<<(n + 255) / 256, 256>>>(n, d_a, d_b, d_c);

  hipMemcpy(c, d_c, sizeof(float) * n, hipMemcpyDefault);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
